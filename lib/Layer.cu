#include "hip/hip_runtime.h"
#include "cldl/Layer.h"

#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <ctgmath>
#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <fstream>
#include <iostream>
#include <math.h>
#include <fstream>
#include <iostream>
#include <string>
#include <numeric>
#include <vector>
#include <fstream>

#define MAX_BLOCKSIZE 1024


// GPU FUNCTIONS //

__global__ void gpu_setLearningRate(Neuron* n, double _learningRate, int nNeurons) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<nNeurons)
        device_setLearningRate(&n[i], _learningRate);
}

__global__ void gpu_setInputs(Neuron* n, double *list, int nNeurons) {
    int i = threadIdx.x; // Input index
    int j = (blockIdx.x*blockDim.y) + threadIdx.y; // Neuron index
    if(j < nNeurons)
        n[j].inputs[i] = list[i];
}

__global__ void gpu_setErrorCoeff(Neuron *n, double _globalCoeff, double _backwardsCoeff,
                                  double _midCoeff, double _forwardCoeff,
                                 double _localCoeff, double _echoCoeff, int nNeurons) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<nNeurons) {
        *n[i].backwardsCoeff = _backwardsCoeff;
        *n[i].midCoeff = _midCoeff;
        *n[i].forwardCoeff = _forwardCoeff;
        *n[i].globalCoeff = _globalCoeff;
        *n[i].localCoeff = _localCoeff;
        *n[i].echoCoeff = _echoCoeff;
    }
}

__global__ void gpu_setWeights(Neuron* n, double *list, int nNeurons) {
    int i = threadIdx.x; // Input index
    int j = (blockIdx.x*blockDim.y) + threadIdx.y; // Neuron index
    if(j < nNeurons)
        n[j].weights[i] = list[i];
}

__global__ void gpu_setBackwardError(Neuron*n, double _leadBackwardError, int nNeurons) {
    double leadBackwardError = _leadBackwardError;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<nNeurons)
        device_setBackwardError(leadBackwardError, &n[i]);
}

__global__ void gpu_calcErrorWeightProductSum(Neuron* n, int nNeurons, int nInputs, double* sumlist) {
    int i = threadIdx.x; // Input index
    int j = (blockIdx.x*blockDim.y) + threadIdx.y; // Neuron index

    if(i < nInputs && j < nNeurons)
        n[j].ErrorWeightProducts[i] = n[j].weights[i] * (*n[j].backwardError);
    __syncthreads();

    if (j == 0) {
        double sum = 0.0;
        for (int a = 0; a < nNeurons; a++) {
            sum += n[a].ErrorWeightProducts[i];
        }
        sumlist[i] = sum;
    }
}

/*__global__ void gpu_setForwardError(Neuron*n, double _leadForwardError) {
    int i = threadIdx.x;
    *n[i].forwardError = _leadForwardError;
}*/

__global__ void gpu_calcOutputs(Neuron* neurons, int* layerHasReported){
    device_calcOutput(&neurons[blockIdx.x]);
    __syncthreads();
    device_calcOutputCont(&neurons[blockIdx.x], layerHasReported);
    __syncthreads();
}

__global__ void gpu_propErrorBackwards(Neuron *n, double* _sumList, int nNeurons) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double* sumList = _sumList;
    if (i<nNeurons)
        device_propErrorBackward(sumList[i], &n[i]);
}

__global__ void gpu_updateWeights(Neuron *n, int nNeurons){
    int i = threadIdx.x;    //Input index
    int j = (blockIdx.x*blockDim.y) + threadIdx.y;  //Neuron index
    //double force = 1;
    if (j<nNeurons) {
        n[j].weights[i] += (*n[j].learningRate) * n[j].inputs[i] * (*n[j].backwardError); // * force;
    }
}

__global__ void gpu_getOutputs(Neuron* n, double* _outputs, int nNeurons){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<nNeurons) {
        _outputs[i] = *n[i].output;
    }
}

// HOST FUNCTIONS //

__host__ Layer::Layer(int _nNeurons, int _nInputs){
    nNeurons = _nNeurons; // number of neurons in this layer
    nInputs = _nInputs; // number of inputs to each neuron

    neurons = (Neuron*) (malloc(sizeof(Neuron) * nNeurons));
    for (int i=0; i<nNeurons; i++){
        Neuron* j = new Neuron(nInputs);
        neurons[i] = *j;
    }

    hipMalloc((void**) &gpu_sumlist, sizeof(double)*_nInputs);
    hipMalloc((void**) &gpu_weights, sizeof(double)*nInputs);
    hipMalloc( (void**) &gpu_inputs, sizeof(double)*nInputs);
    hipMalloc( (void**) &gpu_neurons, sizeof(Neuron)*nNeurons);
    hipMemcpy(gpu_neurons, neurons, sizeof(Neuron)*nNeurons, hipMemcpyHostToDevice);
}

__host__ Layer::~Layer(){
    for(int i=0;i<nNeurons;i++) {
        neurons[i].~Neuron();
    }
    free(neurons);
    hipFree(gpu_inputs);
    hipFree(gpu_neurons);
}

//*************************************************************************************
//initialisation:
//*************************************************************************************

__host__ void Layer::initLayer(int _layerIndex, Neuron::weightInitMethod _wim, Neuron::biasInitMethod _bim, Neuron::actMethod _am){
    myLayerIndex = _layerIndex;
    for (int i=0; i<nNeurons; i++){
        neurons[i].initNeuron(i, myLayerIndex, _wim, _bim, _am);
    }
}

__host__ void Layer::setlearningRate(double _learningRate){
    int B = std::ceil(float(nNeurons)/MAX_BLOCKSIZE);   // Total number of blocks required
    int T = MAX_BLOCKSIZE;
    if (nNeurons<MAX_BLOCKSIZE){
        T = nNeurons;
    }
    //printf("%d, %d\n", B, T);
    learningRate=_learningRate;
    gpu_setLearningRate<<<B,T>>>(gpu_neurons, learningRate, nNeurons);
    hipDeviceSynchronize();
}

__host__ void Layer::setErrorCoeff(double _globalCoeff, double _backwardsCoeff,
                            double _midCoeff, double _forwardCoeff,
                            double _localCoeff, double  _echoCoeff) {
    int B = std::ceil(float(nNeurons)/MAX_BLOCKSIZE);   // Total number of blocks required
    int T = MAX_BLOCKSIZE;
    if (nNeurons<MAX_BLOCKSIZE){
        T = nNeurons;
    }
    gpu_setErrorCoeff<<<B,T>>>(gpu_neurons, _globalCoeff, _backwardsCoeff,
                                      _midCoeff, _forwardCoeff, _localCoeff, _echoCoeff, nNeurons);
    hipDeviceSynchronize();
}

//this method is for testing only
__host__ void Layer::setWeights(double* _weightsList) {
    hipMemcpy(gpu_weights, _weightsList, sizeof(double)*nInputs,hipMemcpyHostToDevice);
    int nThreads = nInputs * nNeurons;          // Total number of CUDA threads required
    int blockYDim = MAX_BLOCKSIZE/nInputs;      // Size of a block's Y dimension
    int blockSize = nInputs * blockYDim;        // Size of required block
    int B = std::ceil(float(nThreads)/blockSize);   // Total number of blocks required
    dim3 T = dim3(nInputs, blockYDim);          // 2D block dimensions
    gpu_setWeights<<<B,T>>>(gpu_neurons, gpu_weights, nNeurons);
}

//*************************************************************************************
//forward propagation of inputs:
//*************************************************************************************

__host__ void Layer::setInputs(double *_inputs) {
    inputs = _inputs;
    hipMemcpy(gpu_inputs, inputs, sizeof(double)*nInputs,hipMemcpyHostToDevice);

    int nThreads = nInputs * nNeurons;          // Total number of CUDA threads required
    int blockYDim = MAX_BLOCKSIZE/nInputs;      // Size of a block's Y dimension
    int blockSize = nInputs * blockYDim;        // Size of required block
    int B = std::ceil(float(nThreads)/blockSize);   // Total number of blocks required
    dim3 T = dim3(nInputs, blockYDim);          // 2D block dimensions
    gpu_setInputs<<<B,T>>>(gpu_neurons, gpu_inputs, nNeurons);

    hipDeviceSynchronize();
}

__host__ void Layer::propInputs(double* _gpu_InputOutputs) {
    int nThreads = nInputs * nNeurons;          // Total number of CUDA threads required
    int blockYDim = MAX_BLOCKSIZE/nInputs;      // Size of a block's Y dimension
    int blockSize = nInputs * blockYDim;        // Size of required block
    int B = std::ceil(float(nThreads)/blockSize);   // Total number of blocks required
    dim3 T = dim3(nInputs, blockYDim);          // 2D block dimensions
    gpu_setInputs<<<B,T>>>(gpu_neurons, _gpu_InputOutputs, nNeurons);
    hipDeviceSynchronize();
}

__host__ void Layer::calcOutputs(){
    // block id gets neuron
    int* _layerHasReported;
    gpu_allocateInt(&_layerHasReported, 0);
    hipMemcpy(_layerHasReported, &layerHasReported, sizeof(int), hipMemcpyHostToDevice);

    gpu_calcOutputs<<<nNeurons, 1>>>(gpu_neurons, _layerHasReported);
    hipDeviceSynchronize();

    hipMemcpy(&layerHasReported, _layerHasReported, sizeof(int), hipMemcpyDeviceToHost);
}

//*************************************************************************************
//forward propagation of error:
//*************************************************************************************

/*__host__ void Layer::setForwardError(double _leadForwardError){
    this is only for the first layer
    leadForwardError=_leadForwardError;
    gpu_setForwardError<<<1,nNeurons>>>(gpu_neurons, leadForwardError);
    hipDeviceSynchronize();
}*/

//__host__ void Layer::propErrorForward(int _index, double _value){
//    for (int i=0; i<nNeurons; i++){
//        neurons[i]->propErrorForward(_index, _value);
//    }
//}

/*__host__ double Layer::getForwardError(int _neuronIndex){
    return (neurons[_neuronIndex].getForwardError());
}*/

//*************************************************************************************
//back propagation of error:
//*************************************************************************************

__host__ void Layer::setBackwardError(double _leadBackwardError) {
    leadBackwardError = _leadBackwardError;
    int B = std::ceil(float(nNeurons)/MAX_BLOCKSIZE);   // Total number of blocks required
    int T = MAX_BLOCKSIZE;
    if (nNeurons<MAX_BLOCKSIZE){
        T = nNeurons;
    }
    //printf("%d, %d\n", B, T);
    gpu_setBackwardError<<<B,T>>>(gpu_neurons, leadBackwardError, nNeurons);
    hipDeviceSynchronize();
}

__host__ double* Layer::calcErrorWeightProductSum() {
    int nThreads = nInputs * nNeurons;          // Total number of CUDA threads required
    int blockYDim = MAX_BLOCKSIZE/nInputs;      // Size of a block's Y dimension
    int blockSize = nInputs * blockYDim;        // Size of required block
    int B = std::ceil(float(nThreads)/blockSize);   // Total number of blocks required
    dim3 T = dim3(nInputs, blockYDim);          // 2D block dimensions
    //printf("%d, %d, %d\n", B, nInputs, blockYDim);
    gpu_calcErrorWeightProductSum<<<B,T>>>(gpu_neurons, nNeurons, nInputs, gpu_sumlist);
    hipDeviceSynchronize();
    return gpu_sumlist;
}

__host__ void Layer::propErrorBackward(double* _sumList) {
    int B = std::ceil(float(nNeurons)/MAX_BLOCKSIZE);   // Total number of blocks required
    int T = MAX_BLOCKSIZE;
    if (nNeurons<MAX_BLOCKSIZE){
        T = nNeurons;
    }
    //printf("%d, %d\n", B, T);
    gpu_propErrorBackwards<<<B,T>>>(gpu_neurons, _sumList, nNeurons);
    hipDeviceSynchronize();
}

//*************************************************************************************
//learning:
//*************************************************************************************

__host__ void Layer::updateWeights() {
    int nThreads = nInputs * nNeurons;          // Total number of CUDA threads required
    int blockYDim = MAX_BLOCKSIZE/nInputs;      // Size of a block's Y dimension
    int blockSize = nInputs * blockYDim;        // Size of required block
    int B = std::ceil(float(nThreads)/blockSize);   // Total number of blocks required
    dim3 T = dim3(nInputs, blockYDim);          // 2D block dimensions

    gpu_updateWeights<<<B,T>>>(gpu_neurons, nNeurons);
    hipDeviceSynchronize();
}

//*************************************************************************************
//getters:
//*************************************************************************************

__host__ Neuron* Layer::getNeuron(int _neuronIndex){
    return (&neurons[_neuronIndex]);
}

__host__ int Layer::getnNeurons(){
    return (nNeurons);
}

__host__ double* Layer::getOutputs(){
    double* _outputs;
    hipMalloc(&_outputs, sizeof(double)*nNeurons);
    int B = std::ceil(float(nNeurons)/MAX_BLOCKSIZE);   // Total number of blocks required
    int T = MAX_BLOCKSIZE;
    if (nNeurons<MAX_BLOCKSIZE){
        T = nNeurons;
    }
    gpu_getOutputs<<<B,T>>>(gpu_neurons, _outputs, nNeurons);
    return _outputs;
}

__host__ double Layer::getOutput(int _neuronIndex) {
    return (neurons[_neuronIndex].getOutput());
}

__host__ double Layer::getErrorWeightProductSum(int index) {
    double _sum = 0.0;
    double* sum = gpu_sumlist + index;
    hipMemcpy(&_sum, sum, sizeof(double), hipMemcpyDeviceToHost);
    return _sum;
}

__host__ double Layer::getBackwardError(int _neuronIndex){
    return (neurons[_neuronIndex].getBackwardError());
}

__host__ void Layer::printWeights(FILE* weights) {
    for (int i=0;i<nNeurons;i++) {
        for (int j=0;j<nInputs;j++) {
            fprintf(weights,"%f, ", neurons[i].getWeight(j));
        }
        fprintf(weights,"\n");
    }
    fprintf(weights,"\n");
}

