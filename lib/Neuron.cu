#include "hip/hip_runtime.h"
#include "cldl/Neuron.h"

#include <hip/hip_runtime.h>



//*************************************************************************************
// constructor de-constructor
//*************************************************************************************

__host__ Neuron::Neuron(int _nInputs)
{
    // initialisation
    gpu_allocateInt(&nInputs, _nInputs);
    gpu_allocateInt(&myLayerIndex, 0);
    gpu_allocateInt(&myNeuronIndex, 0);
    hipMalloc((void**)&initialWeights, sizeof(double)*_nInputs);
    gpu_allocateDouble(&learningRate, 0.0);

    gpu_allocateInt(&iHaveReported, 0);

    // forward propagation of inputs
    hipMalloc((void**)&inputs, sizeof(double)*_nInputs);
    gpu_allocateDouble(&bias, 0.0);
    gpu_allocateDouble(&sum, 0.0);
    gpu_allocateDouble(&output, 0.0);

    // forward propagation of error
    hipMalloc((void**)&inputErrors, sizeof(double)*_nInputs);
    gpu_allocateDouble(&forwardError, 0.0);


    // back propagation of error
    gpu_allocateDouble(&backwardError, 0.0);

    // mid propagation of error
    hipMalloc((void**)&inputMidErrors, sizeof(double)*_nInputs);
    gpu_allocateDouble(&midError, 0.0);


    //
    // learning variables
    //
    gpu_allocateDouble(&backwardsCoeff, 0.0);
    gpu_allocateDouble(&midCoeff, 0.0);
    gpu_allocateDouble(&forwardCoeff, 0.0);
    gpu_allocateDouble(&globalCoeff, 0.0);

    hipMalloc((void**)&weights, sizeof(double)*_nInputs);

    gpu_allocateDouble(&weightSum, 0.0);
    gpu_allocateDouble(&maxWeight, 1.0);
    gpu_allocateDouble(&minWeight, 1.0);
    gpu_allocateDouble(&weightChange, 0.0);
    gpu_allocateDouble(&weightsDifference, 0.0);
    gpu_allocateInt(&actMet, 0);

    // global setting
    gpu_allocateDouble(&globalError, 0.0);
    gpu_allocateDouble(&localError, 0.0);
    gpu_allocateDouble(&echoCoeff, 0.0);
    gpu_allocateDouble(&localCoeff, 0.0);

    gpu_allocateDouble(&overallError, 0.0);
    gpu_allocateDouble(&echoError, 0.0);
    hipMalloc((void**)&echoErrors, sizeof(double)*_nInputs);

    //cout << "neuron" << endl;

}

__host__ Neuron::~Neuron(){
    //initialisation
    hipFree(nInputs);
    hipFree(learningRate);
    hipFree(myLayerIndex);
    hipFree(initialWeights);
    hipFree(myNeuronIndex);

    hipFree(iHaveReported);

    // forward propagation of inputs
    hipFree(inputs);
    hipFree(bias);
    hipFree(sum);
    hipFree(output);

    // forward propagation of error
    hipFree(inputErrors);
    hipFree(forwardError);

    // back propagation of error
    hipFree(backwardError);

    // mid propagation of error
    hipFree(inputMidErrors);
    hipFree(midError);


    //learning
    hipFree(backwardsCoeff);
    hipFree(midCoeff);
    hipFree(forwardCoeff);
    hipFree(globalCoeff);
    hipFree(weights);
    hipFree(weightSum);
    hipFree(maxWeight);
    hipFree(minWeight);
    hipFree(weightChange);
    hipFree(weightsDifference);
    hipFree(actMet);

    // global setting
    hipFree(globalError);
    hipFree(localError);
    hipFree(echoCoeff);
    hipFree(localCoeff);

    hipFree(overallError);
    hipFree(echoError);
    hipFree(echoErrors);
}


//*************************************************************************************
//initialisation:
//*************************************************************************************

//TODO test init neuron
__host__ void Neuron::initNeuron(int _neuronIndex, int _layerIndex, weightInitMethod _wim, biasInitMethod _bim, actMethod _am){
    hipMemcpy(myLayerIndex, &_layerIndex, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(myNeuronIndex, &_neuronIndex, sizeof(int), hipMemcpyHostToDevice);
    switch(_wim) {
        case W_ZEROS:
            gpu_setValuesInArray<<<1,getNInputs()>>>(0, weights);
            break;
        case W_ONES:
            gpu_setValuesInArray<<<1,getNInputs()>>>(1, weights);
            break;
        case W_RANDOM:
            //TODO set the random
//            weights[i] = (((double) rand() / (RAND_MAX))); //* 2) -1;
            break;
            //cout << " Neuron: weight is: " << weights[i] << endl;
            /* rand function generates a random function between
             * 0 and RAND_MAX, after the devision the weights are
             * set to a value between 0 and 1 */
    }
    hipMemcpy(initialWeights, weights, sizeof(double)*getNInputs(), hipMemcpyDeviceToDevice);

    gpu_setDouble<<<1,1>>>(weightSum, 0);
    gpu_getSumAndMaxMin<<<1,1>>>(weightSum, maxWeight, minWeight, weights, getNInputs());

    switch (_bim){
        case B_NONE:
            gpu_setDouble<<<1,1>>>(bias, 0.0);
            break;
        case B_RANDOM:
            gpu_setDouble<<<1,1>>>(bias, ((double)rand()/RAND_MAX));
            break;
    }
    switch(_am){
        case Act_Sigmoid:
            gpu_setInt<<<1,1>>>(actMet, 0);
            break;
        case Act_Tanh:
            gpu_setInt<<<1,1>>>(actMet, 1);
            break;
        case Act_NONE:
            gpu_setInt<<<1,1>>>(actMet, 2);
            break;
    }
}

__host__ void Neuron::setLearningRate(double _learningRate){
    gpu_setDouble<<<1,1>>>(learningRate, _learningRate);
}

__host__ double Neuron::getLearningRate() {
    double _learningRate;
    hipMemcpy(&_learningRate, learningRate, sizeof(double), hipMemcpyDeviceToHost);
    return _learningRate;
}


//*************************************************************************************
//forward propagation of inputs:
//*************************************************************************************
__host__ void Neuron::setInput(int _index, double _value) {
    assert((_index>=0)&&(_index<getNInputs()));
    gpu_setValueInArray<<<1,1>>>(_value, _index, inputs);
}

__host__ double Neuron::getInput(int index) {
    double _input = 0.0;
    assert(index < getNInputs());

    double* input = inputs + index;
    hipMemcpy(&_input, input, sizeof(double), hipMemcpyDeviceToHost);
    return _input;
}

__host__ void Neuron::propInputs(int _index,  double _value){
    assert((_index>=0)&&(_index < getNInputs()));
    gpu_setValueInArray<<<1,1>>>(_value,_index, inputs);
}

//TODO calcOutput

//*************************************************************************************
//forward propagation of error:
//*************************************************************************************

__host__ void Neuron::setForwardError(double _value) {
    gpu_setValuesInArray<<<1, getNInputs()>>>(_value, inputErrors);
}

__host__ double Neuron::getInputError(int _index) {
    double _inputError = 0.0;
    assert(_index < getNInputs());

    double* inputError = inputErrors + _index;
    hipMemcpy(&_inputError, inputError, sizeof(double), hipMemcpyDeviceToHost);
    return _inputError;
}

__host__ void Neuron::propErrorForward(int _index, double _value){
    assert((_index>=0)&&(_index<getNInputs()));
    gpu_setValueInArray<<<1,1>>>(_value, _index, inputErrors);
}


//TODO calcForwardError
//__host__ void Neuron::calcForwardError() {
//    double* _value;
//    hipMalloc((void**)&_value, sizeof(double)*getNInputs());
//    gpu_dotProduct<<<1, getNInputs()>>>(inputErrors, weights, _value, fowardError, getNInputs());

    //TODO forwardError must be multiplied with doActivationPrime(sum)
    //TODO assert forwardError isFinite
//}

__host__ double Neuron::getForwardError() {
    double _forwardError = 0.0;
    hipMemcpy(&_forwardError, forwardError, sizeof(double), hipMemcpyDeviceToHost);
    return _forwardError;
}


//*************************************************************************************
//back propagation of error
//*************************************************************************************

//TODO setBackwardError
//__host__ void Neuron::setBackwardError(double _leadError){
//    //TODO use doActivationPrime(sum)
//    gpu_setDouble<<<1,1>>>(backwardError,_leadError*doActivationPrime(sum));
//}


//TODO propErrorBackward make it
//__host__ void Neuron::propErrorBackward(double _nextSum){
//    //TODO use doActivationPrime(sum)
//    gpu_setDouble<<<1,1>>>(backwardError,_leadError*doActivationPrime(sum));
//}

__host__ double Neuron::getBackwardError(){
    double _backwardError = 0.0;
    hipMemcpy(&_backwardError, backwardError, sizeof(double), hipMemcpyDeviceToHost);
    return _backwardError;
}

__host__ double Neuron::getEchoError() {
    double _echoError = 0.0;
    hipMemcpy(&_echoError, echoError, sizeof(double), hipMemcpyDeviceToHost);
    return _echoError;
}
//TODO echoErrorBackward
//__host__ void Neuron::echoErrorBackward(double _nexSum) {
//    //TODO use doActivationPrime(sum)
//    gpu_setDouble<<<1,1>>>(echoError,_nextSum*doActivationPrime(sum));
//}

//*************************************************************************************
//MID propagation of error
//*************************************************************************************

__host__ void Neuron::setMidError(double _leadMidError) {
    gpu_setValuesInArray<<<1, getNInputs()>>>(_leadMidError, inputMidErrors);
}

__host__ double Neuron::getInputMidErrors(int index) {
    double _inputMidError = 0.0;
    assert(index < getNInputs());

    double* inputMidError = inputMidErrors + index;
    hipMemcpy(&_inputMidError, inputMidError, sizeof(double), hipMemcpyDeviceToHost);
    return _inputMidError;
}

__host__ void Neuron::calcMidError() {
    double* _value;
    hipMalloc((void**)&_value, sizeof(double)*getNInputs());
    gpu_dotProduct<<<1, getNInputs()>>>(inputMidErrors, weights, _value, midError, getNInputs());
    // TODO midError with doActivationPrime
}


__host__ double Neuron::getMidError() {
    double _midError = 0.0;
    hipMemcpy(&_midError, backwardError, sizeof(double), hipMemcpyDeviceToHost);
    return _midError;
}

//TODO propMidErrorForward

//TODO propMidErrorBackward

//*************************************************************************************
//exploding/vanishing gradient:
//*************************************************************************************

//TODO getError

//*************************************************************************************
//learning
//*************************************************************************************

//TODO setErrorCoeff

//TODO updateWeights

//*************************************************************************************
//global settings
//*************************************************************************************

//TODO setGlobalError

//TODO getGlobalError

//TODO setEchoError

//TODO echoErrorForward

//TODO calcEchoError

//*************************************************************************************
//local backpropagation of error
//*************************************************************************************

//TODO setLocalError

//TODO propGlobalErrorBackwardLocally

//TODO getLocalError

//*************************************************************************************
// getters:
//*************************************************************************************

__host__ double Neuron::getOutput(){
    double _output=0;
    hipMemcpy(&_output, output, sizeof(double), hipMemcpyDeviceToHost);
    return _output;
}

__host__ double Neuron::getSumOutput(){
    double _sum=0;
    hipMemcpy(&_sum, sum, sizeof(double), hipMemcpyDeviceToHost);
    return _sum;
}

__host__ double Neuron::getMaxWeight(){
    double _maxWeight=0;
    hipMemcpy(&_maxWeight, maxWeight, sizeof(double), hipMemcpyDeviceToHost);
    return _maxWeight;
}

__host__ double Neuron::getMinWeight(){
    double _minWeight=0;
    hipMemcpy(&_minWeight, minWeight, sizeof(double), hipMemcpyDeviceToHost);
    return _minWeight;
}

__host__ double Neuron::getSumWeight(){
    double _weightSum=0;
    hipMemcpy(&_weightSum, weightSum, sizeof(double), hipMemcpyDeviceToHost);
    return _weightSum;
}


//double Neuron::getWeightChange(){
//    weightsDifference = 0;
//    weightChange = 0;
//    for (int i=0; i<nInputs; i++){
//        weightsDifference = weights[i] - initialWeights[i];
//        weightChange += pow(weightsDifference,2);
//    }
//    return (weightChange);
//}

//TODO getWeightDistance

__host__ int Neuron::getNInputs(){
    int _nInputs=0;
    hipMemcpy(&_nInputs, nInputs, sizeof(int), hipMemcpyDeviceToHost);
    return _nInputs;
}


//TODO getWeights

//TODO getInitWeights

//*************************************************************************************
//saving and inspecting
//*************************************************************************************

//TODO saveWeights

//TODO printNeuron

//*************************************************************************************
//helper host functions:
//*************************************************************************************
__host__ void gpu_allocateInt(int** pointer, int value){
    hipMalloc(pointer, sizeof(int));
    gpu_setInt<<<1,1>>>(*pointer, value);
}
__host__ void gpu_allocateDouble(double** pointer, double value){
    hipMalloc(pointer, sizeof(double));
    gpu_setDouble<<<1,1>>>(*pointer, value);
}

//*************************************************************************************
//device CUDA kernels:
//*************************************************************************************
__device__ void device_doActivation(double* output, double _sum, int* actMet) {
    switch(*actMet){
        case 0:
            *output = (1/(1+(exp(-_sum)))) - 0.5;
            break;
        case 1:
            *output = tanh(_sum);
            break;
        case 2:
            *output = _sum;
            break;
    }
}

__device__ void device_doActivationPrime(double* output, double _input, int* actMet){
    switch(*actMet){
        case 0:
            device_doActivation(output, _input, actMet);
            *output = 1 * (0.5 + *output) * (0.5 - *output); //exp(-_input) / pow((exp(-_input) + 1),2);
            break;
        case 1:
            *output = 1 - pow(tanh(_input), 2.0);
            break;
        case 2:
            *output = 1;
            break;
    }
}

//*************************************************************************************
//global CUDA kernels:
//*************************************************************************************

__global__ void gpu_setValuesInArray(double _value, double* list){
    list[threadIdx.x] = _value;
}

__global__ void gpu_setValueInArray(double _value, int index, double* list){
    list[index] = _value;
}

__global__ void gpu_getSumAndMaxMin(double* sum, double* max_list, double* list_min, double* list, int length){
    for (int i=0; i<length; i++){
        *sum = *sum + fabs(list[i]);
        *max_list = max(*max_list, list[i]);
        *list_min = min(*list_min, list[i]);
    }
}


__global__ void gpu_setInt(int* pointer, int value) {
    *pointer = value;
}

__global__ void gpu_setDouble(double* pointer, double value){
    *pointer = value;
}

__global__ void gpu_doActivation(double* output, double _sum, int* actMet) {
    double sum = _sum;
    device_doActivation(output, sum, actMet);
}

__global__ void gpu_doActivationPrime(double* output, double _input, int* actMet) {
    double input = _input;
    device_doActivationPrime(output, input, actMet);
}

__global__ void gpu_dotProduct(double* list1, double* list2, double* _value, double* _target, int arrayLength){
    int idx = threadIdx.x;
    int stride = blockDim.x;

    double target = 0.0;
    for (int i = idx; i < arrayLength; i+=stride){
        target += list1[i]*list2[i];
    }

    _value[idx] = target;
    __syncthreads();

    for (int size = stride/2; size>0; size/=2){
        if (idx < size){
            _value[idx] += _value[idx+size];
        }
        __syncthreads();
    }
    if (idx == 0){
        *_target = _value[0];
    }
}