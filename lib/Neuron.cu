#include "cldl/Neuron.h"

#include <assert.h>
#include <iostream>
#include <ctgmath>
#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <fstream>
#include <iostream>
#include <math.h>
#include <string>
#include <numeric>
#include <vector>
#define CUDA_HOSTDEV __host__ __device__

using namespace std;

//*************************************************************************************
// constructor de-constructor
//*************************************************************************************

__host__ Neuron::Neuron(int _nInputs)
{

    hipMalloc((void**)&nInputs, sizeof(int));
    hipMemcpy(nInputs, &_nInputs, sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&weights, sizeof(double)*_nInputs);
    hipMalloc((void**)&initialWeights, sizeof(double)*_nInputs);
    hipMalloc((void**)&inputs, sizeof(double)*_nInputs);
    hipMalloc((void**)&inputErrors, sizeof(double)*_nInputs);
    hipMalloc((void**)&inputMidErrors, sizeof(double)*_nInputs);
    hipMalloc((void**)&echoErrors, sizeof(double)*_nInputs);

    //cout << "neuron" << endl;

}

__host__ Neuron::~Neuron(){
    hipFree(weights);
    hipFree(initialWeights);
    hipFree(inputs);
    hipFree(inputErrors);
    hipFree(inputMidErrors);
    hipFree(echoErrors);
}

__host__ int Neuron::getNInputs(){
    int _nInputs=0;
    hipMemcpy(&_nInputs, nInputs, sizeof(int), hipMemcpyDeviceToHost);
    return _nInputs;
}
