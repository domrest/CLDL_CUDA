#include "hip/hip_runtime.h"
#include "cldl/Neuron.h"

#include <assert.h>
#include <iostream>
#include <ctgmath>
#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <fstream>
#include <iostream>
#include <math.h>
#include <string>
#include <numeric>
#include <vector>

using namespace std;

//*************************************************************************************
// constructor de-constructor
//*************************************************************************************

__host__ Neuron::Neuron(int _nInputs)
{

    hipMalloc((void**)&nInputs, sizeof(int));
    hipMemcpy(nInputs, &_nInputs, sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&learningRate, sizeof(double));
    hipMemcpyFromSymbol(learningRate, HIP_SYMBOL(0.0), sizeof(double));

    hipMalloc((void**)&weights, sizeof(double)*_nInputs);
    hipMalloc((void**)&initialWeights, sizeof(double)*_nInputs);
    hipMalloc((void**)&inputs, sizeof(double)*_nInputs);
    hipMalloc((void**)&inputErrors, sizeof(double)*_nInputs);
    hipMalloc((void**)&inputMidErrors, sizeof(double)*_nInputs);
    hipMalloc((void**)&echoErrors, sizeof(double)*_nInputs);

    //cout << "neuron" << endl;

}

__host__ Neuron::~Neuron(){
    hipFree(nInputs);
    hipFree(learningRate);

    hipFree(weights);
    hipFree(initialWeights);
    hipFree(inputs);
    hipFree(inputErrors);
    hipFree(inputMidErrors);
    hipFree(echoErrors);
}


//*************************************************************************************
//initialisation:
//*************************************************************************************

//TODO initNeuron

__host__ void Neuron::setLearningRate(double _learningRate){
    hipMemcpy(learningRate, &_learningRate, sizeof(double), hipMemcpyHostToDevice);
}

__host__ double Neuron::getLearningRate() {
    double _learningRate;
    hipMemcpy(&_learningRate, learningRate, sizeof(double), hipMemcpyDeviceToHost);
    return _learningRate;
}


//*************************************************************************************
//forward propagation of inputs:
//*************************************************************************************

//TODO setInput

//TODO propInputs

//TODO calcOutput

//*************************************************************************************
//forward propagation of error:
//*************************************************************************************

__host__ void Neuron::setForwardError(double _value) {
    gpu_setValuesInArray<<<1, getNInputs()>>>(_value, inputErrors);
}

__host__ double Neuron::getInputError(int index) {
    double _inputError = 0.0;
    assert(index < getNInputs());

    double* inputError = inputErrors + index;
    hipMemcpy(&_inputError, inputError, sizeof(double), hipMemcpyDeviceToHost);
    return _inputError;
}

__host__ void Neuron::propErrorForward(int _index, double _value){
    assert((_index>=0)&&(_index<getNInputs()));
    gpu_setValueInArray<<<1,1>>>(_value, _index, inputErrors);
}


//TODO calcForwardError

//TODO getForwardError

//*************************************************************************************
//back propagation of error
//*************************************************************************************

//TODO setBackwardError

//TODO propErrorBackward

//TODO getBackwardError

//TODO getEchoError

//TODO echoErrorBackward

//*************************************************************************************
//MID propagation of error
//*************************************************************************************

//TODO setMidError

//TODO calcMidError

//TODO getMidError

//TODO propMidErrorForward

//TODO propMidErrorBackward

//*************************************************************************************
//exploding/vanishing gradient:
//*************************************************************************************

//TODO getError

//*************************************************************************************
//learning
//*************************************************************************************

//TODO setErrorCoeff

//TODO updateWeights

//TODO doActivation

//TODO doActivationPrime

//*************************************************************************************
//global settings
//*************************************************************************************

//TODO setGlobalError

//TODO getGlobalError

//TODO setEchoError

//TODO echoErrorForward

//TODO calcEchoError

//*************************************************************************************
//local backpropagation of error
//*************************************************************************************

//TODO setLocalError

//TODO propGlobalErrorBackwardLocally

//TODO getLocalError

//*************************************************************************************
// getters:
//*************************************************************************************

//TODO getOutput

//TODO getSumOutput

//TODO getMaxWeight

//TODO getMinWeight

//TODO getSumWeight

//TODO getWeightChange

//TODO getWeightDistance

__host__ int Neuron::getNInputs(){
    int _nInputs=0;
    hipMemcpy(&_nInputs, nInputs, sizeof(int), hipMemcpyDeviceToHost);
    return _nInputs;
}

//TODO getWeights

//TODO getInitWeights

//*************************************************************************************
//saving and inspecting
//*************************************************************************************

//TODO saveWeights

//TODO printNeuron


//*************************************************************************************
//global kernel:
//*************************************************************************************

__global__ static void gpu_setValuesInArray(double _value, double* list){
    list[threadIdx.x] = _value;
}

__global__ static void gpu_setValueInArray(double _value, int index, double* list){
    list[index] = _value;
}
