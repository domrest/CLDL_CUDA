#include "hip/hip_runtime.h"
#include "cldl/Layer.h"
#include "gtest/gtest.h"

using namespace std;

__global__ void checkNInputs(Neuron* n){
    int i = threadIdx.x;
    *n[i].nInputs = 2;
}

TEST(CUDATest, testObjectPointerCallsList){
    Neuron* n;
    n = (Neuron*) (malloc(sizeof(Neuron) * 5));
    for (int i=0; i<5; i++){
        Neuron* j = new Neuron(1);
        n[i] = *j;
    }

    Neuron* d_n;
    hipMalloc((void**) &d_n, sizeof(Neuron)*5);
    hipMemcpy(d_n, n, sizeof(Neuron)*5, hipMemcpyHostToDevice);

    checkNInputs<<<1,5>>>(d_n);
    ASSERT_EQ(n[2].getNInputs(), 2);

}

TEST(LayerTest, testLayerConstructor){
    Layer *l;
    l = new Layer(10, 10);
    //Check there are 10 neurons in the layer
    ASSERT_EQ(l->getnNeurons(), 10);

    //Check that neurons have 10 inputs
    Neuron *n;
    n = l->getNeuron(0);
    ASSERT_EQ(n->getNInputs(), 10);
}

//TODO testLayerDestructor

//TODO testInitLayer

TEST(LayerTest, testLayerSetLearningRate) {
    Layer *l;
    l = new Layer(10, 10);
    l->setlearningRate(0.1);

    Neuron *n;
    n = l->getNeuron(0);
    ASSERT_EQ(n->getLearningRate(), 0.1);
}

TEST(LayerTest, testLayerSetInputs) {
    Layer *l;
    l = new Layer(100, 10);

    double in[10] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0};
    l->setInputs(in);
    ASSERT_EQ(l->inputs[3], 4.0);

    Neuron *n;
    n = l->getNeuron(3);
    ASSERT_EQ(n->getNInputs(), 10);
    ASSERT_EQ(n->getInput(5), 6.0);
    ASSERT_EQ(n->getInput(2), 3.0);

    Neuron *n2;
    n2 = l->getNeuron(99);
    ASSERT_EQ(n2->getNInputs(), 10);
    ASSERT_EQ(n2->getInput(5), 6.0);
    ASSERT_EQ(n2->getInput(2), 3.0);
}

//TODO testLayerPropInputs
TEST(LayerTest, testLayerPropInputs) {
    Layer *l;
    l = new Layer(200, 10);

    double prevLayerOuts[10] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0};
    double* gpu_InputOutputs;
    hipMalloc( (void**) &gpu_InputOutputs, sizeof(double)*10);
    hipMemcpy(gpu_InputOutputs, prevLayerOuts, sizeof(double)*10,hipMemcpyHostToDevice);

    l->propInputs(gpu_InputOutputs);
    //ASSERT_EQ(l->inputs[3], 4.0);
    Neuron *n;
    n = l->getNeuron(3);
    ASSERT_EQ(n->getNInputs(), 10);
    ASSERT_EQ(n->getInput(5), 6.0);
    ASSERT_EQ(n->getInput(2), 3.0);

    Neuron *n2;
    n2 = l->getNeuron(150);
    ASSERT_EQ(n2->getNInputs(), 10);
    ASSERT_EQ(n2->getInput(5), 6.0);
    ASSERT_EQ(n2->getInput(2), 3.0);
}

/*TEST(LayerTest, testLayerSetForwardError) {
    Layer *l;
    l = new Layer(10, 10);
    l->setForwardError(0.1);
    ASSERT_EQ(l->getForwardError(0), 0.1);
}*/

TEST(LayerTest, testLayerSetBackwardError) {
    Layer *l;
    l = new Layer(10, 10);
    l->setBackwardError(0.01);
    ASSERT_EQ(l->leadBackwardError, 0.01);
    ASSERT_EQ(l->getBackwardError(5), 0.0025);
}

TEST(LayerTest, testLayerSetErrorCoeff) {
    Layer *l;
    l = new Layer(10, 10);
    l->setErrorCoeff(0, 1, 0, 0, 0, 0);

    Neuron *n;
    n = l->getNeuron(5);
    ASSERT_EQ(n->getBackwardsCoeff(), 1.0);
}

TEST(LayerTest, testLayerUpdateWeights) {
    Layer *l;
    l = new Layer(10, 10);
    l->setBackwardError(2.0);
    l->setlearningRate(2.0);
    l->setErrorCoeff(0, 1, 0, 0, 0, 0);
    double in[10] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0};
    l->setInputs(in);
    l->updateWeights();

    Neuron *n;
    n = l->getNeuron(5);
    ASSERT_EQ(n->getWeight(0), 1.0);
    ASSERT_EQ(n->getWeight(1), 2.0);
    /* This test doesn't pass if BackwardError
       and LearningRate are set to 0.1.
       It says:
       "getWeight(0) = 0.0025."
       "This does not equal 0.0025" */
}

//TODO testLayerCalcErrorWeightProductSum
TEST(LayerTest, testLayerCalcErrorWeightProductSum) {
    Layer *l;
    l = new Layer(10, 12);
    l->setBackwardError(2.0);
    l->setlearningRate(2.0);
    double in[12] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0, 11.0, 12.0};
    l->setInputs(in);
    l->updateWeights();
    double* sumlist;
    sumlist = l->calcErrorWeightProductSum();

    Neuron *n;
    n = l->getNeuron(2);
    ASSERT_EQ(l->leadBackwardError, 2.0);
    ASSERT_EQ(l->getBackwardError(0), 0.5);
    ASSERT_EQ(n->getWeight(11), 12.0);
    ASSERT_EQ(n->getInput(11), 12.0);

    ASSERT_EQ(n->getErrorWeightProducts(0),0.5);
    ASSERT_EQ(n->getErrorWeightProducts(1),1.0);
    ASSERT_EQ(n->getErrorWeightProducts(11), 6.0);

    ASSERT_EQ(l->getSum(0), 5);
    ASSERT_EQ(l->getSum(1), 10);
    ASSERT_EQ(l->getSum(11), 60);
}

//TODO test_propErrorBackward
TEST(LayerTest, testLayerpropErrorBackwards) {
    //Create "final" layer
    Layer *l;
    l = new Layer(10, 12);
    l->setlearningRate(2.0);
    l->setBackwardError(2.0);
    double in[12] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0, 11.0, 12.0};
    l->setInputs(in);
    l->updateWeights();
    double *sumlist;
    sumlist = l->calcErrorWeightProductSum();

    //Create "previous" layer
    Layer *l2;
    l2 = new Layer(12, 10);
    l2->setBackwardError(2.0);
    l2->setlearningRate(2.0);
    l2->propErrorBackward(sumlist);

    ASSERT_EQ(l2->getnNeurons(), 12);
    ASSERT_EQ(l2->getBackwardError(0), 1.25);
    ASSERT_EQ(l2->getBackwardError(1), 2.5);
    ASSERT_EQ(l2->getBackwardError(11), 15);
}

//TODO testLayerCalcOutputs
