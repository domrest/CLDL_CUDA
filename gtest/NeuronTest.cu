#include "hip/hip_runtime.h"
#include "cldl/Neuron.h"
#include "gtest/gtest.h"
#include <hip/hip_runtime.h>

using namespace std;


TEST(NeuronTest, testNeuronIntialisationAndNInputs){
    Neuron *n;
    n = new Neuron(1);
    ASSERT_EQ(n->getNInputs(), 1);
}



TEST(NeuronTest, testSetLearningRate){
    Neuron *n;
    n = new Neuron(1);
    n->setLearningRate(2.0);
    ASSERT_EQ(n->getLearningRate(), 2.0);
}

TEST(NeuronTest, testSetInputErrors){
    Neuron *n;
    n = new Neuron(4);
    n->setForwardError(2.0);
    n->propErrorForward(2, 4.0);
    ASSERT_EQ(n->getInputError(1), 2.0);
    ASSERT_EQ(n->getInputError(2), 4.0);

}


TEST(NeuronTest, testSumAndMaxMin){
    double *sum, *d_sum, *max, *d_max, *min, *d_min, *list, *d_list;

    gpu_allocateDouble(&d_sum, 0.0);
    gpu_allocateDouble(&d_max, 1.0);
    gpu_allocateDouble(&d_min, 1.0);


    hipMalloc((void**)&d_list, sizeof(double)*4);

    list = new double[4];
    list[0] = 0.5;
    list[1] = 1.0;
    list[2] = 1.5;
    list[3] = 2.0;


    sum = (double*)malloc(sizeof(double));
    max = (double*)malloc(sizeof(double));
    min = (double*)malloc(sizeof(double));

    hipMemcpy(d_list, list, sizeof(double)*4, hipMemcpyHostToDevice);

    gpu_getSumAndMaxMin<<<1,1>>>(d_sum, d_max, d_min, d_list, 4);
    hipMemcpy(sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(max, d_max, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(min, d_min, sizeof(double), hipMemcpyDeviceToHost);

    ASSERT_EQ(*sum, 5.0);
    ASSERT_EQ(*max, 2.0);
    ASSERT_EQ(*min, 0.5);

}


int main(int argc, char** argv){
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
