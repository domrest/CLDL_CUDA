#include "hip/hip_runtime.h"
#include "gtest/gtest.h"

__global__ void test1(float *n){
    *n = *n * 4.0;
}

TEST(CUDATest, testfloat){
    float *a, *d_a;
    a = (float*)malloc(sizeof(float));
    *a = 2.0f;
    hipMalloc((void**)&d_a,sizeof(float));
    hipMemcpy(d_a, a, sizeof(float), hipMemcpyHostToDevice);

    test1<<<1,1>>>(d_a);
    hipMemcpy(a, d_a, sizeof(float), hipMemcpyDeviceToHost);
    ASSERT_EQ(8.0f, *a);
}

//int main(){
//
//
//}
